#include "hip/hip_runtime.h"
﻿// Implementacja mno�enia macierzy jest realizowana za pomoca typowego 
// algorytmu podr�cznikowego. 
#include <stdio.h>
#include <time.h>
#include <windows.h>
#include "omp.h"

#define USE_MULTIPLE_THREADS true
#define MAXTHREADS 128
int NumThreads;
double start;

static const int ROWS = 1024;     // liczba wierszy macierzy
static const int COLUMNS = 1024;  // lizba kolumn macierzy

float matrix_a[ROWS][COLUMNS];    // lewy operand 
float matrix_b[ROWS][COLUMNS];    // prawy operand
float matrix_r[ROWS][COLUMNS];    // wynik


void initialize_matrices()
{
    // zdefiniowanie zawarosci poczatkowej macierzy
//#pragma omp parallel for 
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLUMNS; j++) {
            matrix_a[i][j] = (float)rand() / RAND_MAX;
            matrix_b[i][j] = (float)rand() / RAND_MAX;
            matrix_r[i][j] = 0.0;
        }
    }
}

void initialize_matricesZ()
{
    // zdefiniowanie zawartosci poczatkowej macierzy
#pragma omp parallel for 
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLUMNS; j++) {
            matrix_r[i][j] = 0.0;
        }
    }
}


void multiply_matrices_IJK()
{
    // mnozenie macierzy 
#pragma omp parallel for 
    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLUMNS; j++) {
            float sum = 0.0;
            for (int k = 0; k < COLUMNS; k++) {
                sum = sum + matrix_a[i][k] * matrix_b[k][j];
            }
            matrix_r[i][j] = sum;
        }
    }
}

void multiply_matrices_IKJ()
{
    // mnozenie macierzy 
#pragma omp parallel for 
    for (int i = 0; i < ROWS; i++)
        for (int k = 0; k < COLUMNS; k++)
            for (int j = 0; j < COLUMNS; j++)
                matrix_r[i][j] += matrix_a[i][k] * matrix_b[k][j];

}

void multiply_matrices_JIK()
{
    // mnozenie macierzy 
#pragma omp parallel for 

    for (int j = 0; j < COLUMNS; j++) {
        for (int i = 0; i < ROWS; i++) {
            float sum = 0.0;
            for (int k = 0; k < COLUMNS; k++) {
                sum = sum + matrix_a[i][k] * matrix_b[k][j];
            }
            matrix_r[i][j] = sum;
        }
    }
}
void multiply_matrices_JKI()
{
    // mnozenie macierzy 
#pragma omp parallel for 
    for (int j = 0; j < COLUMNS; j++)
        for (int k = 0; k < COLUMNS; k++)
            for (int i = 0; i < ROWS; i++)
                matrix_r[i][j] += matrix_a[i][k] * matrix_b[k][j];

}


void print_elapsed_time()
{
    double elapsed;
    double resolution;

    // wyznaczenie i zapisanie czasu przetwarzania
    elapsed = (double)clock() / CLK_TCK;
    resolution = 1.0 / CLK_TCK;
    printf("Czas: %8.4f sec \n",
        elapsed - start);

}

int main(int argc, char* argv[])
{
    //	 start = (double) clock() / CLK_TCK ;

    //Determine the number of threads to use
    if (USE_MULTIPLE_THREADS) {
        SYSTEM_INFO SysInfo;
        GetSystemInfo(&SysInfo);
        NumThreads = SysInfo.dwNumberOfProcessors;
        if (NumThreads > MAXTHREADS)
            NumThreads = MAXTHREADS;
    }
    else
        NumThreads = 1;
    printf("liczba watkow  = %d\n\n", NumThreads);

    initialize_matrices();
    start = (double)clock() / CLK_TCK;
    multiply_matrices_IJK();
    printf("IJK ");
    print_elapsed_time();
    initialize_matricesZ();
    start = (double)clock() / CLK_TCK;
    multiply_matrices_IKJ();
    printf("IKJ ");
    print_elapsed_time();
    initialize_matricesZ();
    start = (double)clock() / CLK_TCK;
    multiply_matrices_JIK();
    printf("JIK ");
    print_elapsed_time();
    initialize_matricesZ();
    start = (double)clock() / CLK_TCK;
    multiply_matrices_JKI();
    printf("JKI ");
    print_elapsed_time();

    return(0);
}